#include "hip/hip_runtime.h"
#include "crypt/crypt.cuh"
#include "utils.h"
#include "wordgen.h"
#include <algorithm>
#include <iostream>
#include <map>
#include <math.h>
// #include <mpi.h>
#include <omp.h>
#include <set>
#include <signal.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <string>
#include <vector>

// #include "crypt/crypt_r.h"

typedef unsigned long long ull;
typedef long long ll;

#define WAIT_TIME 100

int stop = 0;
int num_cifras = 0;
std::set<int> falta = std::set<int>();
int mpi_rank = 0;
int mpi_size = 1;
// MPI_Comm comm = MPI_COMM_WORLD;
int falta_size = 0;

// void mpi_master_relay() {
//   std::set<int> done;
//   int next_done;
//   MPI_Request request;
//   MPI_Status status;

//   fprintf(stderr, "P%d iniciando mpi_master_relay\n", mpi_rank);

//   // Receber notificação de cifra K quebrada,
//   //   broadcast de K para todos workers.
//   while (!stop) {
//     int flag;
//     MPI_Irecv(&next_done, 1, MPI_INT, MPI_ANY_SOURCE, 0, comm, &request);
//     sleep_for(WAIT_TIME);
//     MPI_Test(&request, &flag, &status);
//     if (flag) {
//       done.insert(next_done);

// // Processar a lista para que o master também retire os prontos
// #pragma omp critical(falta_global)
//       {
//         if (falta.count(next_done) > 0)
//           falta.erase(next_done);
//         falta_size = falta.size();
//       }

//       fprintf(stderr, "P%d removendo cifra %d\n", mpi_rank, next_done);

//       // Replicar para os workers
//       MPI_Bcast(&next_done, 1, MPI_INT, 0, comm);
//     }
//   }
// }

// void mpi_worker_listener() {
//   // Seção de sincronização de progresso
//   std::set<int> done;
//   int next_done;
//   MPI_Request request;
//   MPI_Status status;

//   fprintf(stderr, "P%d iniciando mpi_worker_listener\n", mpi_rank);

//   while (falta.size() > 0 && !stop) {
//     int flag;
//     // Receber int K do broadcast do root
//     //   adicionar esse int K no set done.
//     MPI_Ibcast(&next_done, 1, MPI_INT, 0, comm, &request);
//     sleep_for(WAIT_TIME);
//     MPI_Test(&request, &flag, &status);
//     if (flag) {
// // Processar a lista para que o master também retire os prontos
// #pragma omp critical(falta_global)
//       {
//         if (falta.count(next_done) > 0)
//           falta.erase(next_done);
//         falta_size = falta.size();
//       }
//       fprintf(stderr, "P%d removendo cifra %d\n", mpi_rank, next_done);
//     }
//   }
// }

void force_stop(int signal) {
  fprintf(stderr, "Encerramento forçado: sinal %d\n", signal);
  stop = 1;
}

__global__ void cuda_do_des(char *cifra, bool *resolvido, char *solucao,
                            int *sal_por_cifra, int num_cifras, char *sal,
                            int num_sais, ll inicio, int intervalo,
                            int run_count) {
  int thread = threadIdx.x;
  ll start = threadIdx.x + inicio;
  ll step = blockDim.x * intervalo;

  printf("GPU thread %d starting at %lld with step %lld!\n", thread, start,
         step);

  crypt_des_data meu_crypt_data;

  char senha[16] = "\0\0\0\0\0\0\0\0\0\0\0\0\0\0\0";
  char *result;
  char *ssenha;
  char *cifrados = (char *)malloc(sizeof(char) * 16 * num_sais);

  ll vetor[8] = {start, -1, -1, -1, -1, -1, -1, -1};
  static int maxSize = 64;

  while (run_count--) {
    // Atualizar nova senha, corrigindo a base do número
    int overflow, pos = 0;
    while ((overflow = (vetor[pos] / maxSize)) > 0 && pos < 8) {
      vetor[pos] = vetor[pos] % maxSize;
      // senha[pos] = ascii64[vetor[pos]];
      vetor[++pos] += overflow;
    }
    // senha[pos] = ascii64[vetor[pos]];

    senha[0] = bit7[vetor[0] + 1];
    senha[1] = bit7[vetor[1] + 1];
    senha[2] = bit7[vetor[2] + 1];
    senha[3] = bit7[vetor[3] + 1];
    senha[4] = bit7[vetor[4] + 1];
    senha[5] = bit7[vetor[5] + 1];
    senha[6] = bit7[vetor[6] + 1];
    senha[7] = bit7[vetor[7] + 1];

    for (int i = 0; i < num_sais; i++) {
      int desloc = i * 16;
      result = crypt_des_cuda(senha, &sal[desloc], &meu_crypt_data);

      // Copiar para o vetor
      for (int kkk = 0; kkk < 13; kkk++) {
        cifrados[desloc + kkk] = result[kkk];
      }
      cifrados[desloc + 13] = 0;
      // printf("%s + %s -> %s\n", senha, &sal[desloc], &cifrados[desloc]);
    }

    for (int i = 0; i < num_cifras; i++) {
      if (!resolvido[i]) {
        ssenha = &cifrados[sal_por_cifra[i] * 16];
        if (strncmp_cuda(ssenha, &cifra[i * 16], 13) == 0) {
          resolvido[i] = true;
          solucao[i * 16 + 0] = senha[0];
          solucao[i * 16 + 1] = senha[1];
          solucao[i * 16 + 2] = senha[2];
          solucao[i * 16 + 3] = senha[3];
          solucao[i * 16 + 4] = senha[4];
          solucao[i * 16 + 5] = senha[5];
          solucao[i * 16 + 6] = senha[6];
          solucao[i * 16 + 7] = senha[7];
          solucao[i * 16 + 8] = '\0';

          printf("%s %s\n", &cifra[i * 16], senha);
        }
      }
    }

    // Passar para próxima senha
    vetor[pos] += step;
  }
}

int main(int argc, char *argv[]) {
  signal(SIGINT, force_stop);

  // int thread_level;
  // MPI_Init_thread(&argc, &argv, MPI_THREAD_MULTIPLE, &thread_level);

  // mpi_rank = 0;
  // mpi_size = 1;
  // MPI_Comm comm = MPI_COMM_WORLD;
  // MPI_Comm_rank(comm, &mpi_rank);
  // MPI_Comm_size(comm, &mpi_size);

  // Obter comprimento máximo
  int comprimento = 0;
  ll maximo = 64L;
  if (argc == 2) {
    sscanf(argv[1], "%d", &comprimento);
    comprimento = MIN(8, comprimento);
    for (int i = 1; i < comprimento; i++) {
      maximo++;
      maximo *= (ll)maxSize;
    }
  } else {
    fprintf(stderr, "Falta argumento: %s <comprimento_maximo> [início]\n",
            argv[0]);
    fprintf(stderr, "Uso: Informe pela entrada padrão o número de cifras, "
                    "número de threads  e em "
                    "seguida digite\n");
    fprintf(stderr, "     as cifras uma por linha.\n");
    exit(1);
  }

  // Ler senhas e sincronizar com outros processos MPI
  num_cifras = 0;
  std::map<std::string, int> sais;
  std::set<std::string> sais_vistos;
  std::map<int, std::string> sal_por_cifra;
  std::vector<int> sal_por_indice;
  char **cifras;
  char *cbloco;
  if (mpi_rank == 0) {
    // ROOT
    std::cin >> num_cifras;
    // MPI_Bcast(&num_cifras, 1, MPI_INT, 0, comm);
    getchar();
    sal_por_indice = std::vector<int>(num_cifras);
    cifras = new char *[num_cifras];
    cbloco = new char[num_cifras * 32];
    std::string cifra;
    std::vector<std::string> vec_cifras;
    for (int i = 0; i < num_cifras; i++) {
      getline(std::cin, cifra);
      falta.insert(i);
      vec_cifras.push_back(cifra);
    }
    std::sort(vec_cifras.begin(), vec_cifras.end());
    for (int i = 0; i < num_cifras; i++) {
      cifras[i] = &cbloco[i * 32];
      strncpy(cifras[i], vec_cifras[i].data(), 16);
      if (!sais.count(vec_cifras[i].substr(0, 2))) {
        sais_vistos.insert(vec_cifras[i].substr(0, 2));
        sais[vec_cifras[i].substr(0, 2)] = sais.size();
      }
      sal_por_cifra[i] = vec_cifras[i].substr(0, 2);
      sal_por_indice[i] = sais[vec_cifras[i].substr(0, 2)];
    }
    // MPI_Bcast(cifras[0], num_cifras * 32, MPI_CHAR, 0, comm);
    // } else {
    // // Not root
    // MPI_Bcast(&num_cifras, 1, MPI_INT, 0, comm);
    // cifras = new char *[num_cifras];
    // cbloco = new char[num_cifras * 32];
    // sal_por_indice = std::vector<int>(num_cifras);
    // MPI_Bcast(cbloco, num_cifras * 32, MPI_CHAR, 0, comm);
    // for (int i = 0; i < num_cifras; i++) {
    //   cifras[i] = &cbloco[i * 32];
    //   std::string cifra(&cbloco[i * 32], &cbloco[i * 32 + 16]);
    //   falta.insert(i);
    //   if (!sais.count(cifra.substr(0, 2))) {
    //     sais_vistos.insert(cifra.substr(0, 2));
    //     sais[cifra.substr(0, 2)] = sais.size();
    //   }
    //   sal_por_cifra[i] = cifra.substr(0, 2);
    //   sal_por_indice[i] = sais[cifra.substr(0, 2)];
    // }
  }

  // falta_size = falta.size();

  int num_sais = sais.size();
  char *todos_sais = new char[16 * num_sais];
  int sad = 0;
  for (auto &sal : sais) {
    // std::cout << sal.first << " e " << sal.second << "\n";
    // strncpy(&todos_sais[16 * sad], sal.first.data(), 4);
    sal.first.copy(&todos_sais[16 * sad], sal.first.size() + 1);
    todos_sais[16 * sad + sal.first.size()] = '\0';
    sad++;
  }

  for (int i = 0; i < num_sais; i++) {
    printf("%s\n", &todos_sais[i * 16]);
    fflush(stdout);
  }
  printf("\n");
  fflush(stdout);

  char *g_cifra, *g_solucao, *g_sal;
  hipMalloc((void **)&g_cifra, sizeof(char) * 16 * num_cifras);
  hipMalloc((void **)&g_solucao, sizeof(char) * 16 * num_cifras);
  hipMalloc((void **)&g_sal, sizeof(char) * 16 * num_sais);

  hipMemcpy(g_cifra, cifras[0], sizeof(char) * 16 * num_cifras,
             hipMemcpyHostToDevice);
  hipMemcpy(g_sal, todos_sais, sizeof(char) * 16 * num_sais,
             hipMemcpyHostToDevice);

  bool *g_resolvido;
  hipMalloc((void **)&g_resolvido, sizeof(bool) * num_cifras);

  int *g_sal_por_cifra;
  hipMalloc((void **)&g_sal_por_cifra, sizeof(int) * num_cifras);
  hipMemcpy(g_sal_por_cifra, sal_por_indice.data(), sizeof(int) * num_cifras,
             hipMemcpyHostToDevice);

  dim3 grid(1);
  dim3 blok(64);

  cuda_do_des<<<grid, blok>>>(g_cifra, g_resolvido, g_solucao, g_sal_por_cifra,
                              num_cifras, g_sal, num_sais, 0ll, 1, 266305);

  //   // Iniciar thread de sincronização entre MPI workers
  //   std::thread *sync_thread;
  //   if (mpi_size > 1) {
  //     if (mpi_rank == 0)
  //       sync_thread = new std::thread(mpi_master_relay);
  //     else {
  //       sync_thread = new std::thread(mpi_worker_listener);
  //     }
  //   }

  //   // Usar todos threads disponíveis
  //   int num_threads = omp_get_max_threads();
  //   // num_threads = 1;
  //   omp_set_num_threads(num_threads);
  //   fprintf(stderr, "p%d Usando %d threads\n", mpi_rank, num_threads);

  //   std::map<std::string, std::string> solucoes;
  //   ll i = 0L, counter = 0;
  // #pragma omp parallel reduction(+ : counter)
  //   {
  //     // Inicializar sais (aceleração grande)
  //     std::map<std::string, crypt_data> crypt_data_por_sal;
  //     // std::map<std::string, crypt_des_data> crypt_data_por_sal;
  //     for (auto &ss : sais) {
  //       crypt_data_por_sal[ss] = crypt_data();
  //       // crypt_data_por_sal[ss] = crypt_des_data();
  //     }
  //     crypt_data *crypt_pointer;
  //     // crypt_des_data *crypt_pointer;
  //     char *result;
  //     int thread_rank = omp_get_thread_num();
  //     int inicio = (mpi_rank * omp_get_num_threads()) +
  //     omp_get_thread_num(); int passo = mpi_size * omp_get_num_threads();
  //     fprintf(stderr, "p%d t%d inicia em %d (passo %d), existem %d
  //     threads\n",
  //             mpi_rank, thread_rank, inicio, passo, omp_get_num_threads());
  //     Senha senha(inicio);
  //     ll thread_i;
  //     std::set<int> thread_falta(falta);
  //     int thread_falta_size = num_cifras;
  //     for (thread_i = inicio; thread_i < maximo && !stop; thread_i +=
  //     passo) {
  //       // if ((falta.size() < thread_falta.size())) {
  //       if (falta_size < thread_falta_size) {

  // #pragma omp critical(falta_global)
  //         thread_falta = falta;

  //         thread_falta_size = thread_falta.size();
  //       }
  //       for (auto &e : thread_falta) {
  //         // printf("p%d t%d %s %s\n", mpi_rank, thread_rank, cifras[e],
  //         //        senha.getSenha());
  //         crypt_pointer = &(crypt_data_por_sal[sais[e]]);
  //         result = crypt_r(senha.getSenha(), cifras[e], crypt_pointer);
  //         // result = crypt_des(senha.getSenha(), cifras[e],
  //         crypt_pointer); int ok = strncmp(result, cifras[e], 14) == 0;

  //         if (ok) {
  //           // printf("p%*d, t%*d @ %2.f%%: %s = %s\n",
  //           // (int)ceil(log10(mpi_size)),
  //           //        mpi_rank, (int)ceil(log10(passo)), thread_rank,
  //           //        (thread_i / (double)maximo) * 100, cifras[e],
  //           //        senha.getSenha());
  //           printf("%s %s\n", cifras[e], senha.getSenha());
  //           fflush(stdout);
  //           // solucoes[cifras[e]] = senha.getSenha();
  //           int next_done = e;

  //           if (mpi_rank == 0 || mpi_size == 1) {
  // #pragma omp critical(falta_global)
  //             if (falta.count(e) > 0) {
  //               falta.erase(e);
  //             }

  //             // Replicar para os workers
  //             MPI_Bcast(&next_done, 1, MPI_INT, 0, comm);
  //           } else {
  //             MPI_Send(&next_done, 1, MPI_INT, 0, 0, comm);
  //           }
  //         }
  //       }

  //       if (((thread_i + 1) % 50000) == 0) {
  //         fprintf(stderr, "Realizado %2.f%% ou %llu de %llu\n",
  //                 (thread_i / (double)maximo) * 100, thread_i + 1, maximo);
  //       }

  //       senha.prox(passo);
  //       counter++;
  //     }
  // #pragma omp barrier
  //   }
  stop = true;
  //   if (mpi_size > 1) {
  //     sync_thread->join();
  //     delete sync_thread;
  //   }
  // fprintf(stderr, "[%d] terminou em %llu iterações!!!!\n", mpi_rank,
  // counter); for (auto &e : solucoes) {
  //   printf("%s %s\n", e.first.data(), e.second.data());
  // }

  //   MPI_Finalize();
  hipDeviceSynchronize();

  return 0;
}

#include "hip/hip_runtime.h"
#include "matrix.h"
#include <pthread.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

enum Device { CPU, GPU };

double wtime() {
  struct timeval t;
  gettimeofday(&t, NULL);
  return t.tv_sec + t.tv_usec / 1000000.0;
}

int main(int argc, char **argv) {
  char oper[256], device[256];
  double start_time, end_time;
  // Inicializar para perder o warning (-Wmaybe-uninitialized)
  start_time = wtime();
  int width;

  if ((argc != 4)) {
    printf("Uso: %s <operacao> <width> <cpu|gpu>\n", argv[0]);
    printf("   operacao: [sum, mult, sort]\n");
    exit(EXIT_FAILURE);
  }

  strncpy(oper, argv[1], 256);
  width = atoi(argv[2]);
  strncpy(device, argv[3], 256);

  if (strcmp(device, "cpu") && strcmp(device, "gpu")) {
    printf("Dispositivo inválido. Use \"cpu\" ou \"gpu\".\n");
    exit(EXIT_FAILURE);
  }

  int dev = CPU;
  if (strcmp(device, "gpu") == 0) {
    dev = GPU;
  }

  // Do something
  matrix_t *a = (matrix_t *)matrix_create(width, width);
  matrix_t *ret = (matrix_t *)matrix_create(width, width);
  matrix_randfill(a);

  if (strcmp(oper, "sum") == 0) {
    matrix_t *b = (matrix_t *)matrix_create(width, width);
    matrix_randfill(b);
    start_time = wtime();
    if (dev == GPU)
      matrix_sum_gpu(a, b, ret);
    else
      matrix_sum(a, b, ret);
  }

  if (strcmp(oper, "mult") == 0) {
    matrix_t *b = (matrix_t *)matrix_create(width, width);
    matrix_randfill(b);
    start_time = wtime();
    if (dev == GPU)
      matrix_multiply_gpu(a, b, ret);
    else
      matrix_multiply(a, b, ret);
  }

  if (strcmp(oper, "sort") == 0) {
    // memcpy(ret->data, a->data, sizeof(double *) * a->rows);
    // memcpy(ret->data[0], a->data[0], sizeof(double) * a->rows * a->cols);
    start_time = wtime();
    // if (dev == GPU)
    //   matrix_sort_gpu(a, ret);
    // else
    matrix_sort(a, a);
  }
  // END Do something

  end_time = wtime();

  // matrix_print(ret);
  printf("R[%*d,%*d] = %0.5f\n", (int)ceil(log10(width)), 0,
         (int)ceil(log10(width)), 0, ret->data[0][0]);
  printf("R[%*d,%*d] = %0.5f\n", (int)ceil(log10(width)), width - 1,
         (int)ceil(log10(width)), width - 1, ret->data[width - 1][width - 1]);

  // printf("%d %d %f\n", width, width, end_time - start_time);
  printf("%f\n", end_time - start_time);
  fflush(stdout);

  hipDeviceSynchronize();

  return EXIT_SUCCESS;
}

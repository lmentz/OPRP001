#include "hip/hip_runtime.h"
#include "matrix.h"
#include "utils.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

matrix_t *matrix_create(int rows, int cols) {
  matrix_t *ret = NULL;
  double *bloco = NULL;
  int i = 0;

  ret = (matrix_t *)malloc(sizeof(matrix_t));
  bloco = (double *)malloc(sizeof(double) * rows * cols);

  ret->data = (double **)malloc(sizeof(double *) * rows);
  ret->cols = cols;
  ret->rows = rows;

  for (i = 0; i < rows; i++) {
    ret->data[i] = &bloco[i * cols];
  }

  return ret;
}

void matrix_destroy(matrix_t *m) {
  int i;

  free(m->data[0]);
  for (i = 0; i < m->rows; i++) {
    m->data[i] = NULL;
  }

  free(m->data);
  m->data = NULL;

  m->rows = 0;
  m->cols = 0;

  free(m);
  m = NULL;

  return;
}

void matrix_randfill(matrix_t *m) {
  int i, j;
  for (i = 0; i < m->rows; i++) {
    for (j = 0; j < m->cols; j++) {
      m->data[i][j] = random();
    }
  }
}

void matrix_fill(matrix_t *m, double val) {
  int i, j;
  for (i = 0; i < m->rows; i++) {
    for (j = 0; j < m->cols; j++) {
      m->data[i][j] = val;
    }
  }
}

matrix_t *matrix_multiply(matrix_t *A, matrix_t *B, matrix_t *ret) {
  // Checar se a multiplicação é possível
  // if (A->cols != B->rows) {
  //   printf("Matrizes de formato incompativel\n");
  //   return NULL;
  // }

  int i, j, k;
  int newRows = A->rows;
  int newCols = B->cols;

  for (i = 0; i < newRows; i++) {
    for (j = 0; j < newCols; j++) {
      register double sum = 0;
      for (k = 0; k < A->rows; k++) {
        sum += A->data[i][k] * B->data[k][j];
      }
      ret->data[i][j] = sum;
    }
  }

  return ret;
}

matrix_t *matrix_sum(matrix_t *A, matrix_t *B, matrix_t *ret) {
  // Checar se a soma é possível
  // if (A->rows != B->rows || A->cols != B->cols) {
  //   printf("Matrizes de formato incompativel\n");
  //   return NULL;
  // }

  int i;
  int newRows = A->rows;
  int newCols = A->cols;

  for (i = 0; i < newCols * newRows; i++) {
    ret->data[0][i] = A->data[0][i] + B->data[0][i];
  }

  return ret;
}

matrix_t *matrix_sort(matrix_t *A, matrix_t *ret) {
  merge_sort(ret->data[0], A->rows * A->cols);

  return ret;
}

void matrix_print(matrix_t *m) {
  int i, j;
  for (i = 0; i < m->rows; i++) {
    for (j = 0; j < m->cols; j++) {
      printf("%.17f ", m->data[i][j]);
    }
    printf("\n");
  }
  fflush(stdout);
}

// Métodos de GPU

__global__ void matrix_multiply_kernel(double *a, double *b, double *ret,
                                       int ladoMatriz) {
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  if (y < ladoMatriz && x < ladoMatriz) {
    double soma = 0.0;
    for (int i = 0; i < ladoMatriz; i++) {
      soma += a[y * ladoMatriz + i] * b[i * ladoMatriz + x];
    }
    ret[y * ladoMatriz + x] = soma;
  }
}

matrix_t *matrix_multiply_gpu(matrix_t *A, matrix_t *B, matrix_t *ret) {
  // Checar se a multiplicação é possível
  // if (A->cols != B->rows) {
  //   printf("Matrizes de formato incompativel\n");
  //   return NULL;
  // }

  int newRows = A->rows;
  int newCols = B->cols;

  int alloc_mat_size = sizeof(double) * newRows * newCols;
  int width = newRows;

  double *d_a, *d_b, *d_ret;
  hipMalloc((void **)&d_a, alloc_mat_size);
  hipMalloc((void **)&d_b, alloc_mat_size);
  hipMalloc((void **)&d_ret, alloc_mat_size);

  hipMemcpy(d_a, A->data[0], alloc_mat_size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, B->data[0], alloc_mat_size, hipMemcpyHostToDevice);

  dim3 bloco(CUDA_T2, CUDA_T2);
  dim3 grade((int)ceil((double)width / CUDA_T2),
             (int)ceil((double)width / CUDA_T2));

  matrix_multiply_kernel<<<grade, bloco>>>(d_a, d_b, d_ret, newRows);

  hipMemcpy(ret->data[0], d_ret, alloc_mat_size, hipMemcpyDeviceToHost);

  return ret;
}

__global__ void matrix_sum_kernel(double *a, double *b, double *ret,
                                  int num_cells) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < num_cells) {
    ret[i] = a[i] + b[i];
  }
}

matrix_t *matrix_sum_gpu(matrix_t *A, matrix_t *B, matrix_t *ret) {
  // Checar se a soma é possível
  // if (A->rows != B->rows || A->cols != B->cols) {
  //   printf("Matrizes de formato incompativel\n");
  //   return NULL;
  // }

  int newRows = A->rows;
  int newCols = A->cols;

  int alloc_mat_size = sizeof(double) * newRows * newCols;
  int num_cells = newRows * newCols;

  double *d_a, *d_b, *d_ret;
  hipMalloc((void **)&d_a, alloc_mat_size);
  hipMalloc((void **)&d_b, alloc_mat_size);
  hipMalloc((void **)&d_ret, alloc_mat_size);

  hipMemcpy(d_a, A->data[0], alloc_mat_size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, B->data[0], alloc_mat_size, hipMemcpyHostToDevice);

  int grades = (int)ceil((double)num_cells / CUDA_T);

  matrix_sum_kernel<<<grades, CUDA_T>>>(d_a, d_b, d_ret, num_cells);

  hipMemcpy(ret->data[0], ret, alloc_mat_size, hipMemcpyDeviceToHost);

  return ret;
}

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#define N 2047
#define T 1024

__global__ void vecAdd(int *a, int *b, int *c);

int main() {
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    
    // initialize a and b with real values (NOT SHOWN)
    
    int size = N * sizeof(int);
    
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    for (int i = 0; i < N; i++) {
        a[i] = b[i] = i + 1;
        c[i] = 0;
    }

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    vecAdd<<<(int)ceil(N/(double)T),T>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    printf("c[%*d] = %d\n", (int)log10(N+1), 0, c[0]);
    printf("c[%*d] = %d\n", (int)log10(N+1), N-1, c[N-1]);

    exit(0);
}

__global__ void vecAdd(int *a, int *b, int *c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) 
        c[i] = a[i] + b[i];
}